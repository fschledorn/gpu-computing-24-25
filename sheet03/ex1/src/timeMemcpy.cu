#include "chTimer.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

template<typename F>
double time(F callback) {
    chTimerTimestamp start, stop;
    chTimerGetTime(&start);
    const int cIter = 10;
    for (int i = 0; i < cIter; i++) {
        callback();
    }
    chTimerGetTime(&stop);
    return chTimerElapsedTime(&start, &stop) / cIter;
}

int main() {
    double hostToDev, devToPinned, pinnedToDev, devToHost, devToDev;

    auto timeSize = [&](int size) {
        void* hostMem = malloc(size);
        void* pinnedMem, *deviceMem, *deviceMem2;
        hipError_t err = hipHostMalloc(&pinnedMem, size);
        if (err) printf("hipHostMalloc error: %s, size %i\n", hipGetErrorName(err), size);

        err = hipMalloc(&deviceMem, size);
        if (err) printf("hipMalloc error (deviceMem): %s, size %i\n", hipGetErrorName(err), size);

        err = hipMalloc(&deviceMem2, size);
        if (err) printf("hipMalloc error (deviceMem2): %s, size %i\n", hipGetErrorName(err), size);

        memset(hostMem, 0x11, size);

        hostToDev = time([&](){
            hipMemcpy(deviceMem, hostMem, size, hipMemcpyHostToDevice);
            hipDeviceSynchronize();
        });

        devToPinned = time([&](){
            hipMemcpy(pinnedMem, deviceMem, size, hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
        });

        pinnedToDev = time([&](){
            hipMemcpy(deviceMem, pinnedMem, size, hipMemcpyHostToDevice);
            hipDeviceSynchronize();
        });
        devToHost = time([&](){
            hipMemcpy(hostMem, deviceMem, size, hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
        });
        devToDev = time([&](){
            hipMemcpy(deviceMem, deviceMem2, size, hipMemcpyDeviceToDevice);
            hipDeviceSynchronize();
        });

        hipFree(deviceMem);
        hipFree(deviceMem2);
        hipHostFree(pinnedMem);
        free(hostMem);
        hipDeviceSynchronize();
    };

    // do one cold run to prevent cuda startup latencies in the timings
    timeSize(1 << 10);

    for (int i = 10; i <= 30; i += 2) {
        int size = 1 << i;
        timeSize(size);

        printf("%d,%0.4e,%0.4e,%0.4e,%0.4e,%0.4e\n", size, hostToDev, pinnedToDev, devToHost, devToPinned, devToDev);
        fflush(stdout);
    }
}
